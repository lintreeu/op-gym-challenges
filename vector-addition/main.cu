#include "hip/hip_runtime.h"
int main() {
    const int N = 1 << 20;
    const size_t size = N * sizeof(float);

    // Allocate host memory
    std::vector<float> h_A(N, 1.0f);
    std::vector<float> h_B(N, 2.0f);
    std::vector<float> h_C(N);

    // Allocate device memory
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    // Copy inputs to device
    hipMemcpy(d_A, h_A.data(), size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B.data(), size, hipMemcpyHostToDevice);

    // Launch kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    vector_add<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    // Copy result back
    hipMemcpy(h_C.data(), d_C, size, hipMemcpyDeviceToHost);

    // Verify and print some results
    bool correct = true;
    for (int i = 0; i < N; i++) {
        if (fabs(h_C[i] - 3.0f) > 1e-5) {
            correct = false;
            break;
        }
    }

    std::cout << (correct ? "Vector addition PASSED\n" : "Vector addition FAILED\n");

    // Print first 10 results for verification
    std::cout << "Sample results:\n";
    for (int i = 0; i < 10; ++i) {
        std::cout << "C[" << i << "] = " << h_C[i] << std::endl;
    }

    // Clean up
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}