// main.cu
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

// vector_add.cu
__global__ void vector_add(const float* A, const float* B, float* C, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        C[i] = A[i] + B[i];
    }
}