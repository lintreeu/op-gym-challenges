int main() {
    const int M = 2;
    const int N = 4;
    const float eps = 1e-5f;

    std::vector<float> h_x = {1, 2, 3, 4,  4, 3, 2, 1};   // 2x4
    std::vector<float> h_w = {1, 1, 1, 1};
    std::vector<float> h_y(M * N);
    std::vector<float> h_inv_rms(M);

    float *d_x, *d_w, *d_y, *d_inv_rms;
    hipMalloc(&d_x, M * N * sizeof(float));
    hipMalloc(&d_w, N * sizeof(float));
    hipMalloc(&d_y, M * N * sizeof(float));
    hipMalloc(&d_inv_rms, M * sizeof(float));

    hipMemcpy(d_x, h_x.data(), M * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_w, h_w.data(), N * sizeof(float), hipMemcpyHostToDevice);

    rms_norm_forward<<<M, 32>>>(d_x, d_w, d_y, d_inv_rms, N, eps);
    hipDeviceSynchronize();

    hipMemcpy(h_y.data(), d_y, M * N * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(h_inv_rms.data(), d_inv_rms, M * sizeof(float), hipMemcpyDeviceToHost);

    std::cout << "Output Y:\n";
    for (int i = 0; i < M * N; ++i) {
        std::cout << h_y[i] << " ";
        if ((i + 1) % N == 0) std::cout << std::endl;
    }

    std::cout << "Inverse RMS per row:\n";
    for (int i = 0; i < M; ++i)
        std::cout << h_inv_rms[i] << " ";
    std::cout << std::endl;

    hipFree(d_x);
    hipFree(d_w);
    hipFree(d_y);
    hipFree(d_inv_rms);
    return 0;
}